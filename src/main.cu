#include "hip/hip_runtime.h"
#define GPU_RUNNER 1
#include "renderer.h"

#include "lights/directional_light.h"
#include "lights/lights_list.h"

#include "hittables/hittables_list.h"
#include "hittables/sphere.h"
#include "hittables/cube.h"
#include "hittables/plane.h"

#include "camera.h"
#include "material.h"

#include <iostream>
#include <string>

#define WIDTH 1920
#define HEIGHT 1080

#define SAMPLES 5

#define CUDA(f) err = f;\
    if(err != hipSuccess)\
        printf("Cuda Error: %s\n", hipGetErrorString(err))

void writePPM(const char* path, pixel* img, int width, int height);

__global__ void kernel(pixel* image, emissionPixel* emission, int width, int height, Camera* camera, Hittable** world, Light** lights, Material* materials)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;
    
    // hiprandState_t randState;
    // hiprand_init(x + y * width, 0, 0, &randState);

    // -1 / 1
    float u = ((float)x / (float)width ) * 2.0f - 1.0f;
    float v = ((float)y / (float)height) * 2.0f - 1.0f;

    float pixelOffX = 0.5f / width;
    float pixelOffY = 0.5f / height;

    HitColorGlow result;
    for(int i = 0; i < SAMPLES; ++i)
    {
        HitColorGlow sample = AntiAliasing(u, v, pixelOffX, pixelOffY, camera, world, lights, materials /*, &randState */);
        result.color            += glm::clamp(sample.color,    glm::vec3(0.0f), glm::vec3(1.0f));
        result.emission         += glm::clamp(sample.emission, glm::vec3(0.0f), glm::vec3(1.0f));
        result.emissionStrenght += sample.emissionStrenght;
    }
    
    image   [x + y * width].Set(result.color    / glm::vec3(SAMPLES));
    emission[x + y * width].Set(result.emission / glm::vec3(SAMPLES), result.emissionStrenght / SAMPLES);
}

__global__ void initLights(Light** l_lights, Light** d_lights)
{
    if(threadIdx.x > 0 || threadIdx.y > 0)
        return;

    *(l_lights) = new DirectionalLight({ -0.25f, -0.75f, 0.45f  });
    *(d_lights) = new LightsList(l_lights, 1);
}

__global__ void initWorld(Hittable** l_world, Hittable** d_world)
{
    if(threadIdx.x > 0 || threadIdx.y > 0)
        return;
    
    *(l_world)     = new Sphere({  0.0f, -1000.0f, -4.0f }, 1000.0f, 0);
    *(l_world + 1) = new Sphere({  0.0f,  1.0f,    -4.0f }, 1.0f,    1);
    *(l_world + 2) = new Sphere({ -3.0f,  1.0f,    -4.0f }, 1.0f,    2);
    *(l_world + 3) = new Sphere({  3.0f,  1.0f,    -4.0f }, 1.0f,    3);
    // *(l_world + 2) = new Cube  ({ 2.0f,  2.0f, 2.0f }, { 0.5f, 0.5f, 0.5f }, 0);
    // *(l_world + 2) = new Plane ({ 0.0f, -4.5f, 5.0f }, { 0.0f,  -1.0f, 0.0f }, 2);
    *(d_world)     = new HittablesList(l_world, 4);
}

__global__ void cudaFreeList(void** list, void** device_list, int size)
{
    for(int i = 0; i < size; ++i)
        free(list[i]);

    free(device_list);
}

void gaussianBlur(pixel* img, int width, int height, float sigma, int size) {
    if (size % 2 == 0 || size < 3) {
        std::cerr << "La dimensione del kernel deve essere dispari e maggiore di 1." << std::endl;
        return;
    }

    float kernel[size][size];
    float sum = 0.0;

    //calcolo valori del kernel
    for (int x = -size / 2; x <= size / 2; x++) {
        for (int y = -size / 2; y <= size / 2; y++) {
            float value = exp(-(x * x + y * y) / (2 * sigma * sigma));
            kernel[x + size / 2][y + size / 2] = value;
            sum += value;
        }
    }

    //normalizzo il kernel
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            kernel[i][j] /= sum;
        }
    }

    //applico il blur
    pixel* tempImg = (pixel*)malloc(width * height * sizeof(pixel));

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            float sumX = 0.0, sumY = 0.0, sumZ = 0.0;

            for (int k = -size / 2; k <= size / 2; k++) {
                for (int l = -size / 2; l <= size / 2; l++) {
                    int x = min(max(j + k, 0), width - 1);
                    int y = min(max(i + l, 0), height - 1);

                    sumX += img[y * width + x].x * kernel[k + size / 2][l + size / 2];
                    sumY += img[y * width + x].y * kernel[k + size / 2][l + size / 2];
                    sumZ += img[y * width + x].z * kernel[k + size / 2][l + size / 2];
                }
            }

            // Clamping i valori tra 0 e 255
            tempImg[i * width + j].x = (unsigned char)(max(0.0f, min(255.0f, sumX)));
            tempImg[i * width + j].y = (unsigned char)(max(0.0f, min(255.0f, sumY)));
            tempImg[i * width + j].z = (unsigned char)(max(0.0f, min(255.0f, sumZ)));
        }
    }

    // Copiare l'immagine sfocata nell'array originale
    memcpy(img, tempImg, width * height * sizeof(pixel));
    free(tempImg);
}

int main(int argc, char **argv) 
{
    hipDeviceSetLimit(hipLimitStackSize, 65536);

    hipError_t err = hipSuccess;

    // Allocate Texture Memory
	int totalImageBytes = WIDTH * HEIGHT * sizeof(pixel);
	pixel* h_image = (pixel*) malloc(totalImageBytes);
    
	pixel* d_image;
	CUDA(hipMalloc((void**)&d_image, totalImageBytes));
    
    // Setup
    Camera* d_camera;
    {
        Camera* camera = new Camera(60.0f, WIDTH, HEIGHT, 0.01f, 1000.0f);
        CUDA(hipMalloc((void**)&d_camera, sizeof(Camera)));

        CUDA(hipMemcpy(d_camera, camera, sizeof(Camera), hipMemcpyHostToDevice));

        free(camera);
    }

    // Init Lights
    Light** l_lights;
    CUDA(hipMalloc((void**)&l_lights, 1 * sizeof(Light*)));

    Light** d_lights;
    CUDA(hipMalloc((void**)&d_lights, sizeof(LightsList*)));

    initLights<<<1, 1>>>(l_lights, d_lights);

    // Init World
    Hittable** l_world;
    CUDA(hipMalloc((void**)&l_world, 4 * sizeof(Hittable*)));

    Hittable** d_world;
    CUDA(hipMalloc((void**)&d_world, sizeof(HittablesList*)));

    initWorld<<<1, 1>>>(l_world, d_world);

    // Init Materials
    Material* d_materials;
    CUDA(hipMalloc((void**)&d_materials, 4 * sizeof(Material)));

    {
        Material* materials = new Material[4];
        materials[0] = Material{ glm::vec3{ 0.8f, 0.8f, 0.0f }, 0.0f,  0.0f,  0.0f  };
        materials[1] = Material{ glm::vec3{ 0.8f, 0.2f, 0.1f }, 0.08f, 0.02f, 0.0f  };
        materials[2] = Material{ glm::vec3{ 0.8f, 0.8f, 0.8f }, 0.2f,  0.75f, 0.0f  };
        materials[3] = Material{ glm::vec3{ 0.0f, 0.0f, 0.0f }, 0.05f, 0.0f,  1.85f };

        CUDA(hipMemcpy(d_materials, materials, 4 * sizeof(Material), hipMemcpyHostToDevice));
    }
    
    // Raytrace
	dim3 BlockSize(16, 16, 1);
	dim3 GridSize((WIDTH + 15) / 16, (HEIGHT + 15) / 16, 1);

    printf("Kernel size: %d %d %d (%d %d %d)\n", GridSize.x, GridSize.y, GridSize.z, BlockSize.x, BlockSize.y, BlockSize.z);
	kernel<<<GridSize, BlockSize>>>(d_image, WIDTH, HEIGHT, d_camera, d_world, d_lights, d_materials);

    CUDA(hipDeviceSynchronize());

	CUDA(hipMemcpy(h_image, d_image, totalImageBytes, hipMemcpyDeviceToHost));
    
    //blurring
    // gaussianBlur(h_image, WIDTH, HEIGHT, 10.0f, 11);
    
    // Saving and closing
	writePPM("output.ppm", h_image, WIDTH, HEIGHT);

    // Free
    cudaFreeList<<<1, 1>>>((void**)l_lights, (void**)d_lights, 1);
    cudaFreeList<<<1, 1>>>((void**)l_world,  (void**)d_world,  2);

    hipFree(d_materials);

	hipFree(d_image);
	free(h_image);
	return 0;
}

void writePPM(const char* path, pixel* img, int width, int height)
{
	FILE* file = fopen(path, "wb");
	
	if (!file)
	{
		fprintf(stderr, "Failed to open file\n");
		return;
	}
	
	fprintf(file, "P6\n%d %d\n255\n", width, height);
	
	fwrite(img, sizeof(pixel), width * height, file);
	
	fclose(file);
}

