#include "hip/hip_runtime.h"
#define GPU_RUNNER 1
#include "renderer.h"

#include "lights/directional_light.h"
#include "lights/lights_list.h"

#include "hittables/hittables_list.h"
#include "hittables/sphere.h"
#include "hittables/cube.h"
#include "hittables/plane.h"

#include <iostream>
#include <string>

#define WIDTH 720
#define HEIGHT 405

#define SAMPLES 1

void writePPM(const char* path, pixel* img, int width, int height);

__global__ void kernel(pixel* image, int width, int height, Camera camera, Hittable** world, Light** lights, Material* materials)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;
    
    // hiprandState_t randState;
    // hiprand_init(x + y * width, 0, 0, &randState);

    // -1 / 1
    float u = ((float)x / (float)width ) * 2.0f - 1.0f;
    float v = ((float)y / (float)height) * 2.0f - 1.0f;

    float pixelOffX = 0.5f / width;
    float pixelOffY = 0.5f / height;

    glm::vec3 result{ 0.0f, 0.0f, 0.0f };
    for(int i = 0; i < SAMPLES; ++i)
        result += glm::clamp(AntiAliasing(u, v, pixelOffX, pixelOffY, camera, world, lights, materials /*, &randState */), glm::vec3(0.0f), glm::vec3(1.0f));
    
    image[x + y * width].Set(result / glm::vec3(SAMPLES));
}

__global__ void initLights(Light** l_lights, Light** d_lights)
{
    if(threadIdx.x > 0 || threadIdx.y > 0)
        return;

    *(l_lights) = new DirectionalLight({ -0.25f, -0.75f, 0.45f  });
    *(d_lights) = new LightsList(l_lights, 1);
}

__global__ void initWorld(Hittable** l_world, Hittable** d_world)
{
    if(threadIdx.x > 0 || threadIdx.y > 0)
        return;
    
    *(l_world)     = new Sphere({  0.0f, -1000.0f, -4.0f }, 1000.0f, 0);
    *(l_world + 1) = new Sphere({  0.0f,  1.0f,    -4.0f }, 1.0f,    1);
    *(l_world + 2) = new Sphere({ -3.0f,  1.0f,    -4.0f }, 1.0f,    2);
    *(l_world + 3) = new Sphere({  3.0f,  1.0f,    -4.0f }, 1.0f,    3);
    // *(l_world + 2) = new Cube  ({ 2.0f,  2.0f, 2.0f }, { 0.5f, 0.5f, 0.5f }, 0);
    // *(l_world + 2) = new Plane ({ 0.0f, -4.5f, 5.0f }, { 0.0f,  -1.0f, 0.0f }, 2);
    *(d_world)     = new HittablesList(l_world, 4);
}

__global__ void cudaFreeList(void** list, void** device_list, int size)
{
    for(int i = 0; i < size; ++i)
        free(list[i]);

    free(device_list);
}

void gaussianBlur(pixel* img, int width, int height, float sigma, int size) {
    if (size % 2 == 0 || size < 3) {
        std::cerr << "La dimensione del kernel deve essere dispari e maggiore di 1." << std::endl;
        return;
    }

    float kernel[size][size];
    float sum = 0.0;

    //calcolo valori del kernel
    for (int x = -size / 2; x <= size / 2; x++) {
        for (int y = -size / 2; y <= size / 2; y++) {
            float value = exp(-(x * x + y * y) / (2 * sigma * sigma));
            kernel[x + size / 2][y + size / 2] = value;
            sum += value;
        }
    }

    //normalizzo il kernel
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            kernel[i][j] /= sum;
        }
    }

    //applico il blur
    pixel* tempImg = (pixel*)malloc(width * height * sizeof(pixel));

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            float sumX = 0.0, sumY = 0.0, sumZ = 0.0;

            for (int k = -size / 2; k <= size / 2; k++) {
                for (int l = -size / 2; l <= size / 2; l++) {
                    int x = min(max(j + k, 0), width - 1);
                    int y = min(max(i + l, 0), height - 1);

                    sumX += img[y * width + x].x * kernel[k + size / 2][l + size / 2];
                    sumY += img[y * width + x].y * kernel[k + size / 2][l + size / 2];
                    sumZ += img[y * width + x].z * kernel[k + size / 2][l + size / 2];
                }
            }

            // Clamping i valori tra 0 e 255
            tempImg[i * width + j].x = (unsigned char)(max(0.0f, min(255.0f, sumX)));
            tempImg[i * width + j].y = (unsigned char)(max(0.0f, min(255.0f, sumY)));
            tempImg[i * width + j].z = (unsigned char)(max(0.0f, min(255.0f, sumZ)));
        }
    }

    // Copiare l'immagine sfocata nell'array originale
    memcpy(img, tempImg, width * height * sizeof(pixel));
    free(tempImg);
}

int main(int argc, char **argv) 
{
    // Allocate Texture Memory
	int totalImageBytes = WIDTH * HEIGHT * sizeof(pixel);
	pixel* h_image = (pixel*) malloc(totalImageBytes);
    
	pixel* d_image;
	hipMalloc(&d_image, totalImageBytes);

	
    // Setup
    Camera camera(60.0f, WIDTH, HEIGHT, 0.01f, 1000.0f);

    // Init Lights
    Light** l_lights;
    hipMalloc((void**)&l_lights, 1 * sizeof(Light*));

    Light** d_lights;
    hipMalloc((void**)&d_lights, sizeof(LightsList*));

    initLights<<<1, 1>>>(l_lights, d_lights);

    // Init World
    Hittable** l_world;
    hipMalloc((void**)&l_world, 4 * sizeof(Hittable*));

    Hittable** d_world;
    hipMalloc((void**)&d_world, sizeof(HittablesList*));

    initWorld<<<1, 1>>>(l_world, d_world);

    // Init Materials
    Material* d_materials;
    hipMalloc((void**)&d_materials, 4 * sizeof(Material));

    {
        Material* materials = new Material[4];
        materials[0] = Material{ glm::vec3{ 0.8f, 0.8f, 0.0f }, 0.0f,  0.0f,  0.0f  };
        materials[1] = Material{ glm::vec3{ 0.8f, 0.2f, 0.1f }, 0.08f, 0.02f, 0.0f  };
        materials[2] = Material{ glm::vec3{ 0.8f, 0.8f, 0.8f }, 0.2f,  0.75f, 0.0f  };
        materials[3] = Material{ glm::vec3{ 0.0f, 0.0f, 0.0f }, 0.05f, 0.0f,  1.85f };

        hipMemcpy(d_materials, materials, 4 * sizeof(Material), hipMemcpyHostToDevice);
    }
    

    // Raytrace
	dim3 BlockSize(16, 16, 1);
	dim3 GridSize((WIDTH + 15) / 16, (HEIGHT + 15) / 16, 1);

    printf("%u %u %u - %u %u %u\n", BlockSize.x, BlockSize.y, BlockSize.z, GridSize.x, GridSize.y, GridSize.z);

	kernel<<<GridSize, BlockSize>>>(d_image, WIDTH, HEIGHT, camera, d_world, d_lights, d_materials);
	hipMemcpy(h_image, d_image, totalImageBytes, hipMemcpyDeviceToHost);
	
    //blurring
    // gaussianBlur(h_image, WIDTH, HEIGHT, 10.0f, 11);
    
    // Saving and closing
	writePPM("output.ppm", h_image, WIDTH, HEIGHT);

    // Free
    cudaFreeList<<<1, 1>>>((void**)l_lights, (void**)d_lights, 1);
    cudaFreeList<<<1, 1>>>((void**)l_world,  (void**)d_world,  2);

    hipFree(d_materials);

	hipFree(d_image);
	free(h_image);
	return 0;
}

void writePPM(const char* path, pixel* img, int width, int height)
{
	FILE* file = fopen(path, "wb");
	
	if (!file)
	{
		fprintf(stderr, "Failed to open file\n");
		return;
	}
	
	fprintf(file, "P6\n%d %d\n255\n", width, height);
	
	fwrite(img, sizeof(pixel), width * height, file);
	
	fclose(file);
}

